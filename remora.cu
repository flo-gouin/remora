#include "hip/hip_runtime.h"
#include "remora.h"

struct event{
	int64 time;
	unsigned int event_type;
	unsigned int line;
	char function[64];
	char info[128];
};

event *eventTab = (event*) malloc(MAX_EVENTS * sizeof(event));
//event eventTab[MAX_EVENTS];
unsigned int nbEvent = 0;

int64 timerTab[MAX_TIMERS];

void addTime(unsigned int numTimer, int64 time){
	timerTab[numTimer] += time;
}

double getTimer(unsigned int numTimer){
	return timerTab[numTimer]/getTickFrequency()*1000.0;
}

void initTimer(unsigned int numTimer){
	timerTab[numTimer] = 0;
}

//========== Console functionalities
char indentTab[128] = "\0";

void fctIndent(){
	strcat(indentTab,".	");
}

void loopIndent(){
	strcat(indentTab,"(	");
}

void indent(){
	strcat(indentTab,"	");
}

void unIndent(){
	indentTab[strlen(indentTab)-1] = '\0';
	indentTab[strlen(indentTab)-1] = '\0';
}

void unIndent2(){
	indentTab[strlen(indentTab)-1] = '\0';
	indentTab[strlen(indentTab)-1] = '\0';
}
//=========== End of console functionalities

void remoraHalt(){
	free(eventTab);
	nbEvent = 0;
}

void addIter(const char* info,const char* function,const unsigned int line, unsigned char event_type_p){

}

void addEvent(const char* info,const char* function,const unsigned int line, unsigned char event_type_p){
	if(nbEvent < MAX_EVENTS){
		eventTab[nbEvent].time = getTickCount();
		eventTab[nbEvent].event_type = event_type_p;
		eventTab[nbEvent].line = line;
		strcpy(eventTab[nbEvent].function,function);
		strcpy(eventTab[nbEvent].info,info);
		nbEvent++;
	} else {
		printf("Error: max events limit reached!");
		exit(1);
	}
}

double deltaTime(event eventB , event eventA){
	return (eventB.time - eventA.time)/getTickFrequency()*1000.0;
}

double getTime(event event_p){
	return deltaTime(event_p,eventTab[0]);
}

event searchForNextCorrespondingEvent(int numStartEvent){
	event referingEvent = eventTab[numStartEvent];
	unsigned char targetEventType;
	unsigned int depth = 0;
	switch(referingEvent.event_type){
		case (LOOP_IN):
			targetEventType = LOOP_OUT;
			break;
		case (LOOP_NEST_IN):
			targetEventType = LOOP_NEST_OUT;
			break;
		case (FUNCTION_IN):
			targetEventType = FUNCTION_OUT;
			break;
		case (FUNCTION_CUDA_IN):
			targetEventType = FUNCTION_CUDA_OUT;
			break;
		case (MEMORY_TRANSFERT_IN):
			targetEventType = MEMORY_TRANSFERT_OUT;
			break;
	}
	for(int numEvent = numStartEvent +1; numEvent < nbEvent; numEvent++){
		if(eventTab[numEvent].event_type == referingEvent.event_type){
			depth++;
		} else if(eventTab[numEvent].event_type == targetEventType && depth>0){
			depth--;
		} else if(eventTab[numEvent].event_type == targetEventType && depth==0){
			return eventTab[numEvent];
		}
	}
	return referingEvent;
}

unsigned int computeIterations(int numStartEvent){
	unsigned int iterations = 0;
	event referingEvent = eventTab[numStartEvent];

	unsigned char targetEventType;
	unsigned int depth = 0;
	switch(referingEvent.event_type){
		case (LOOP_IN):
			targetEventType = LOOP_OUT;
			break;
		case (LOOP_NEST_IN):
			targetEventType = LOOP_NEST_OUT;
			break;
		}

	for(int numEvent = numStartEvent +1; numEvent < nbEvent; numEvent++){
		if(eventTab[numEvent].event_type == LOOP_NEST_ITER || eventTab[numEvent].event_type == LOOP_ITER){
			iterations++;
		} else if(eventTab[numEvent].event_type == referingEvent.event_type){
			depth++;
		} else if(eventTab[numEvent].event_type == targetEventType && depth>0){
			depth--;
		} else if(eventTab[numEvent].event_type == targetEventType && depth==0){
			return iterations;
		}
	}
	return iterations;
}

void displayCSVevents(){
	unsigned eventCounter = 0;

	for (int numEvent = 0; numEvent<nbEvent; numEvent++){
		event currEvent = eventTab[numEvent];
		switch(currEvent.event_type){
			case LOOP_OUT:
			case LOOP_NEST_OUT:
			case FUNCTION_OUT:
			case REMORA_IN:
			case REMORA_OUT:
			case REM_INFO:
			case LOOP_ITER:
			case LOOP_NEST_ITER:
			case MEMORY_TRANSFERT_OUT:
				continue;
		}

		switch(currEvent.event_type){
			case FUNCTION_IN:
			case LOOP_IN:
			case LOOP_NEST_IN:
			case MEMORY_TRANSFERT_IN:
				printf("%u	%.03lf\n",eventCounter,deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				break;
		}
		eventCounter++;
	}
}

void displayFunctionsStats_light(){
	printf("=== Function statistics ==================================\n");
	printf("  Function  | runtime(ms)\n");

	for (int numEvent = 0; numEvent<nbEvent; numEvent++){
		event currEvent = eventTab[numEvent];

		switch(currEvent.event_type){
			case FUNCTION_IN:
				printf("%.03lf\n",deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				break;
			case FUNCTION_CUDA_IN:
				printf("%.03lf\n",deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				break;
			case MEMORY_TRANSFERT_IN:
				printf("%.03lf\n",deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				break;
		}
	}
}

void displayFunctionsStats(){
	printf("=== Function statistics ==================================\n");
	printf("  Function  | runtime(ms)\n");

	for (int numEvent = 0; numEvent<nbEvent; numEvent++){
		event currEvent = eventTab[numEvent];

		switch(currEvent.event_type){
			case FUNCTION_IN:
				printf("%s: %.03lfms \n",currEvent.function,deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				break;
			case FUNCTION_CUDA_IN:
				printf("[CUDA] %s: %.03lfms \n",currEvent.function,deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				break;
			case MEMORY_TRANSFERT_IN:
				printf("[MEMORY TRANSFERT] %s: %.03lfms \n",currEvent.function,deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				break;
			
		}
	}
}

void sectorDiagram_datas() {
	printf("=== Sector diagram data ==================================\n");
	printf("tin	tout	lvl	name\n");
	unsigned int level = 0;

	double minTime = getTime(eventTab[1]);
	double maxTime = getTime(eventTab[nbEvent-2]);
	double deltaTime = maxTime - minTime;
	unsigned int resolution = 1;

	float maxSectorsSpace = 330.0;
//	float maxSectorsSpace = 330.0 * (deltaTime/1000.0) /46.248 ;

	printf("\\foreach \\time in {%u,%u,%u,...,%u}{\n",resolution,2*resolution,3*resolution,(int)(deltaTime/1000));
	printf("\\pgfmathparse{\\time*%f}\\let\\sectTime\\pgfmathresult;\n",maxSectorsSpace/(deltaTime/1000));
	printf("\\draw[dashed, color=gray!30] (\\sectTime:2) -- (\\sectTime:7.5);\n");
//	printf("\\draw[color=gray] (\\sectTime:8) node {\\time};\n");
	printf("}\n");

	printf("\\foreach \\time in {%u,%u,%u,...,%u}{\n",resolution,2*resolution,3*resolution,(int)(deltaTime/1000));
	printf("\\pgfmathparse{\\time*%f}\\let\\sectTime\\pgfmathresult;\n",maxSectorsSpace/(deltaTime/1000));
	printf("\\draw[color=gray] (\\sectTime:8) node {\\time};\n");
	printf("}\n");


	printf("\\draw (0:2) -- (0:8);\n");
	printf("\\draw (0:8) node[right] {0s};\n");
	printf("\\draw (%.01f:2) -- (%.01f:8);\n",maxSectorsSpace,maxSectorsSpace);
	printf("\\draw (%.01f:8) node[below right] {%.03fs};\n",maxSectorsSpace,deltaTime/1000.0);

	for (int numLevel = 0; numLevel <= 20; numLevel++) {
		printf("\\sectionLevel{%u}\n",numLevel+2);
		printf("\\draw (0:%.01f) node[below] {%u};\n",(float)(numLevel+2)+0.2,numLevel);
		unsigned int nbElement = 0;
		for (int numEvent = 0; numEvent < nbEvent; numEvent++) {
			event currEvent = eventTab[numEvent];

			if (level == numLevel) {
				nbElement++;
				switch (currEvent.event_type) {
				case LOOP_ITER:
				case LOOP_NEST_ITER:
					printf("\\sectionIter{%.03lf}{%u}\n",(getTime(currEvent)-minTime)/maxTime*maxSectorsSpace,level+2-1);
					break;
				case FUNCTION_IN:
					printf("\\sectionElt{%.03lf}{%.03lf}{%u}{$%s$}{gray!40}\n", (getTime(currEvent)-minTime)/maxTime*maxSectorsSpace,
							(getTime(searchForNextCorrespondingEvent(numEvent))-minTime)/maxTime*maxSectorsSpace,
							level+2, currEvent.function);
					break;
				case FUNCTION_CUDA_IN:
					printf("\\sectionElt{%.03lf}{%.03lf}{%u}{$%s$}{green!75}\n", (getTime(currEvent)-minTime)/maxTime*maxSectorsSpace,
							(getTime(searchForNextCorrespondingEvent(numEvent))-minTime)/maxTime*maxSectorsSpace,
							level+2, currEvent.function);
					break;
				case MEMORY_TRANSFERT_IN:
					printf("\\sectionElt{%.03lf}{%.03lf}{%u}{$%s$}{orange!50}\n", (getTime(currEvent)-minTime)/maxTime*maxSectorsSpace,
							(getTime(searchForNextCorrespondingEvent(numEvent))-minTime)/maxTime*maxSectorsSpace,
							level+2, currEvent.function);
					break;
				case LOOP_IN:
				case LOOP_NEST_IN:
					printf("\\sectionElt{%.03lf}{%.03lf}{%u}{$%s$}{blue!40}\n", (getTime(currEvent)-minTime)/maxTime*maxSectorsSpace,
							(getTime(searchForNextCorrespondingEvent(numEvent))-minTime)/maxTime*maxSectorsSpace,
							level+2, currEvent.info);
					break;
				}
			}
			switch (currEvent.event_type) {
			case FUNCTION_IN:
			case FUNCTION_CUDA_IN:
			case LOOP_IN:
			case LOOP_NEST_IN:
				level++;
				break;
			case FUNCTION_OUT:
			case FUNCTION_CUDA_OUT:
			case LOOP_OUT:
			case LOOP_NEST_OUT:
				level--;
				break;
			}
		}

		if (nbElement == 0)
			break;
	}
}

void displayEvents(){
	event stopEv = eventTab[nbEvent-1];

	printf("%u event(s) recorded during %.03lfms\n",nbEvent,getTime(stopEv));
	printf("#event | runtime(ms) | id |  line | event\n");
	printf("-------+-------------+----+-------+--------\n");

	for (int numEvent = 0; numEvent<nbEvent; numEvent++){
		event currEvent = eventTab[numEvent];
/*		switch(currEvent.event_type){
			case LOOP_OUT:
			case LOOP_NEST_OUT:
			case FUNCTION_OUT:
				unIndent2();
				continue;
			case LOOP_NEST_ITER:
				continue;
		}
*/
		printf("%6i | %11.03lf | %2u | %5u |",numEvent,getTime(currEvent),currEvent.event_type, currEvent.line);
		switch(currEvent.event_type){
			case REMORA_IN:
				printf("**Remora starts collecting metrics**\n");break;
			case REMORA_OUT:
				printf("**End of metrics collection by Remora**\n");break;
			case FUNCTION_IN:
				printf("%s>>%s (%.03lfms)\n",indentTab,currEvent.function,deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				fctIndent();
				break;
			case FUNCTION_OUT:
				unIndent2();
				printf("%s<<%s\n",indentTab,currEvent.function);
				break;
			case FUNCTION_CUDA_IN:
				printf("%s>>[CUDA] %s (%.03lfms)\n",indentTab,currEvent.function,deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				fctIndent();
				break;
			case FUNCTION_CUDA_OUT:
				unIndent2();
				printf("%s<<%s\n",indentTab,currEvent.function);
				break;
			case LOOP_IN:
				printf("%s+loop: %s [%u] (%.03lfms)\n",indentTab,currEvent.info,computeIterations(numEvent),deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				loopIndent();
				break;
			case LOOP_OUT:
				unIndent2();
				printf("%s+loop: %s [%u]\n",indentTab,currEvent.info,computeIterations(numEvent));
				break;
			case LOOP_NEST_IN:
				printf("%s+loop nest: %s [%u] (%.03lfms)\n",indentTab,currEvent.info,computeIterations(numEvent),deltaTime(searchForNextCorrespondingEvent(numEvent),currEvent));
				loopIndent();
				break;
			case LOOP_NEST_OUT:
				unIndent2();
				printf("%s+loop: %s [%u]\n",indentTab,currEvent.info,computeIterations(numEvent));
				break;
			case LOOP_ITER:
				unIndent2();
				printf("%s+--------------%s\n",indentTab,currEvent.info);
				loopIndent();
				break;
			case REM_INFO:
				printf("%s[%s]\n",indentTab,currEvent.info);
				break;
			case ARRAY_INIT:
				printf("%sinit: %s[]\n",indentTab,currEvent.info);
				break;
			case ARRAY_ALLOC:
				printf("%sallocate: %s[]\n",indentTab,currEvent.info);
				break;
			case ARRAY_REDEFINED:
				printf("%spointer redefined: %s[]\n",indentTab,currEvent.info);
				break;
			case ARRAY_FREE:
				printf("%sMemory freed: %s[]\n",indentTab,currEvent.info);
				break;
			default:
				printf("%s%s\n",indentTab,currEvent.info);break;
		}
	}
}

void displayLatexTable(){
	printf("=== Latex Table Data Formated ==================================\n");

	printf("\\begin{longtable}{| c | | >{\\itshape\\scriptsize}c | >{\\itshape\\scriptsize}c | >{\\bfseries\\small}c |}\n");
	printf("\\hline\n");
	printf("\\textbf{Description} & \\textbf{Début} & \\textbf{Fin} & \\textbf{Durée}\\\\\n");
	printf("& \\emph{\\color{darkgray}(s)} & \\emph{\\color{darkgray}(s)} & \\emph{\\color{darkgray}(s)}\\\\\n");

	unsigned int level = 0;

	double minTime = getTime(eventTab[1]);
	double maxTime = getTime(eventTab[nbEvent-2]);
	
	for (int numLevel = 0; numLevel <= 20; numLevel++) {
		printf("\\hline\n");
		printf("\\multicolumn{4}{c}{\\color{darkgray}\\emph{niveau %i}}\\\\\n",numLevel);
		printf("\\hline\n");
	
		unsigned int nbElement = 0;
		for (int numEvent = 0; numEvent < nbEvent; numEvent++) {
			event currEvent = eventTab[numEvent];

			float tin, tout;
			if (level == numLevel) {
				nbElement++;
				switch (currEvent.event_type) {
				case LOOP_ITER:
				case LOOP_NEST_ITER:
					printf("\\hline\n");
					break;
				case FUNCTION_IN:
					tin = (getTime(currEvent)-minTime)/1000.0;
					tout = (getTime(searchForNextCorrespondingEvent(numEvent))-minTime)/1000.0;
					printf("%s & %.03lf & %.03lf & %.03lf \\\\\n",currEvent.function,tin,tout,tout-tin);
					break;
				case FUNCTION_CUDA_IN:
					tin = (getTime(currEvent)-minTime)/1000.0;
					tout = (getTime(searchForNextCorrespondingEvent(numEvent))-minTime)/1000.0;
					printf("%s & \\color{green!75}%.03lf & \\color{green!75}%.03lf & \\color{green!75}%.03lf \\\\\n",currEvent.function,tin,tout,tout-tin);
					break;
				case MEMORY_TRANSFERT_IN:
					tin = (getTime(currEvent)-minTime)/1000.0;
					tout = (getTime(searchForNextCorrespondingEvent(numEvent))-minTime)/1000.0;
					printf("%s & \\color{orange!50}%.03lf & \\color{orange!50}%.03lf & \\color{orange!50}%.03lf \\\\\n",currEvent.function,tin,tout,tout-tin);
					break;
				case LOOP_IN:
				case LOOP_NEST_IN:
					tin = (getTime(currEvent)-minTime)/1000.0;
					tout = (getTime(searchForNextCorrespondingEvent(numEvent))-minTime)/1000.0;
					printf("$l_{elt}$ & \\color{blue!40}%.03lf & \\color{blue!40}%.03lf & \\color{blue!40}%.03lf \\\\\n",tin,tout,tout-tin);
					break;
				}
			}
			switch (currEvent.event_type) {
			case FUNCTION_IN:
			case FUNCTION_CUDA_IN:
			case LOOP_IN:
			case LOOP_NEST_IN:
				level++;
				break;
			case FUNCTION_OUT:
			case FUNCTION_CUDA_OUT:
			case LOOP_OUT:
			case LOOP_NEST_OUT:
				level--;
				break;
			}
		}

		if (nbElement == 0)
			break;
	}
	
	printf("\\hline\n");
	printf("\\caption{\\label{orig_times_exp}Temps d'exécution de l'algorithme simpleflow original sur la Tegra X1}\\\\\n");
	printf("\\end{longtable}\n");
}

void saveResults(const char* filename){
	printf("Writing file: %s\n",filename);

	FILE* file = fopen(filename,"wb");

	if(file == NULL){
		perror("Error accessing file");
		return;
	}

	size_t fileSize = fwrite(eventTab,sizeof(*eventTab),nbEvent,file);
	fclose(file);

	printf("%lu/%u data successfully written in file %s.\n",fileSize,nbEvent,filename);
}
